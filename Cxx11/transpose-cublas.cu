#include "hip/hip_runtime.h"
///
/// Copyright (c) 2013, Intel Corporation
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions
/// are met:
///
/// * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
/// * Redistributions in binary form must reproduce the above
///       copyright notice, this list of conditions and the following
///       disclaimer in the documentation and/or other materials provided
///       with the distribution.
/// * Neither the name of <COPYRIGHT HOLDER> nor the names of its
///       contributors may be used to endorse or promote products
///       derived from this software without specific prior written
///       permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
/// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
/// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
/// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
/// COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
/// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
/// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
/// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
/// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
/// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
/// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.

//////////////////////////////////////////////////////////////////////
///
/// NAME:    transpose
///
/// PURPOSE: This program measures the time for the transpose of a
///          column-major stored matrix into a row-major stored matrix.
///
/// USAGE:   Program input is the matrix order and the number of times to
///          repeat the operation:
///
///          transpose <matrix_size> <# iterations> [tile size]
///
///          An optional parameter specifies the tile size used to divide the
///          individual matrix blocks for improved cache and TLB performance.
///
///          The output consists of diagnostics to make sure the
///          transpose worked and timing statistics.
///
/// HISTORY: Written by  Rob Van der Wijngaart, February 2009.
///          Converted to C++11 by Jeff Hammond, February 2016 and May 2017.
///
//////////////////////////////////////////////////////////////////////

#include "prk_util.h"
#include "prk_cuda.h"

int main(int argc, char * argv[])
{
  std::cout << "Parallel Research Kernels version " << PRKVERSION << std::endl;
  std::cout << "C++11/CUDA Matrix transpose: B = A^T" << std::endl;

  //prk::CUDAinfo();

  //////////////////////////////////////////////////////////////////////
  /// Read and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  int order;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <matrix order>";
      }

      // number of times to do the transpose
      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      // order of a the matrix
      order = std::atol(argv[2]);
      if (order <= 0) {
        throw "ERROR: Matrix Order must be greater than 0";
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Matrix order          = " << order << std::endl;
  std::cout << "Number of iterations  = " << iterations << std::endl;

  //////////////////////////////////////////////////////////////////////
  /// Setup CUDA environment
  //////////////////////////////////////////////////////////////////////

  hipblasHandle_t h;
  hipblasCreate(&h);

  //////////////////////////////////////////////////////////////////////
  /// Allocate space for the input and transpose matrix
  //////////////////////////////////////////////////////////////////////

  const size_t nelems = (size_t)order * (size_t)order;
  const size_t bytes = nelems * sizeof(float);
  float * h_a;
  float * h_b;
  prk::CUDAcheck( hipHostMalloc((float**)&h_a, bytes) );
  prk::CUDAcheck( hipHostMalloc((float**)&h_b, bytes) );

  // fill A with the sequence 0 to order^2-1 as floats
  for (auto j=0; j<order; j++) {
    for (auto i=0; i<order; i++) {
      h_a[j*order+i] = order*j+i;
      h_b[j*order+i] = 0.0f;
    }
  }

  // copy input from host to device
  float * d_a;
  float * d_b;
  prk::CUDAcheck( hipMalloc((float**)&d_a, bytes) );
  prk::CUDAcheck( hipMalloc((float**)&d_b, bytes) );
  prk::CUDAcheck( hipMemcpy(d_a, &(h_a[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDAcheck( hipMemcpy(d_b, &(h_b[0]), bytes, hipMemcpyHostToDevice) );

#if 1
  // We need a vector of ones because CUBLAS saxpy do does
  // correctly implement incx=0.
  float * h_o;
  prk::CUDAcheck( hipHostMalloc((float**)&h_o, bytes) );
  for (auto j=0; j<order; j++) {
    for (auto i=0; i<order; i++) {
      h_o[j*order+i] = 1.0f;
    }
  }
  float * d_o;
  prk::CUDAcheck( hipMalloc((float**)&d_o, bytes) );
  prk::CUDAcheck( hipMemcpy(d_o, &(h_o[0]), bytes, hipMemcpyHostToDevice) );
#endif

#ifdef USE_HOST_BUFFERS
  float p_a = h_a;
  float p_b = h_b;
  float p_o = h_o;
#else
  float * p_a = d_a;
  float * p_b = d_b;
  float * p_o = d_o;
#endif

  auto trans_time = 0.0;

  for (auto iter = 0; iter<=iterations; iter++) {

    if (iter==1) trans_time = prk::wtime();

    float one(1);
    // B += trans(A) i.e. B = trans(A) + B
    hipblasSgeam(h,
                HIPBLAS_OP_T, HIPBLAS_OP_N,   // opA, opB
                order, order,               // m, n
                &one, p_a, order,           // alpha, A, lda
                &one, p_b, order,           // beta, B, ldb
                p_b, order);                // C, ldc (in-place for B)
    // A += 1.0 i.e. A = 1.0 * 1.0 + A
#if 0
    // THIS IS BUGGY
    hipblasSaxpy(h,
                order*order,                // n
                &one,                       // alpha
                &one, 0,                    // x, incx
                p_a, 1);                    // y, incy
#else
    // THIS IS CORRECT
    hipblasSaxpy(h,
                order*order,                // n
                &one,                       // alpha
                p_o, 1,                     // x, incx
                p_a, 1);                    // y, incy
#endif
    // (Host buffer version)
    // The performance is ~10% better if this is done every iteration,
    // instead of only once before the timer is stopped.
    prk::CUDAcheck( hipDeviceSynchronize() );
  }
  trans_time = prk::wtime() - trans_time;

  // copy output back to host
  prk::CUDAcheck( hipMemcpy(&(h_b[0]), d_b, bytes, hipMemcpyDeviceToHost) );

#if 1
  prk::CUDAcheck( hipFree(d_o) );
  prk::CUDAcheck( hipHostFree(h_o) );
#endif

  prk::CUDAcheck( hipFree(d_b) );
  prk::CUDAcheck( hipFree(d_a) );
  prk::CUDAcheck( hipHostFree(h_a) );

  //////////////////////////////////////////////////////////////////////
  /// Analyze and output results
  //////////////////////////////////////////////////////////////////////

  // TODO: replace with std::generate, std::accumulate, or similar
  const auto addit = (iterations+1.) * (iterations/2.);
  auto abserr = 0.0;
  for (auto j=0; j<order; j++) {
    for (auto i=0; i<order; i++) {
      const size_t ij = (size_t)i*(size_t)order+(size_t)j;
      const size_t ji = (size_t)j*(size_t)order+(size_t)i;
      const float reference = static_cast<float>(ij)*(1.+iterations)+addit;
      abserr += std::fabs(h_b[ji] - reference);
    }
  }

#ifdef VERBOSE
  std::cout << "Sum of absolute differences: " << abserr << std::endl;
#endif

  const auto epsilon = 1.0e-8;
  if (abserr < epsilon) {
    std::cout << "Solution validates" << std::endl;
    auto avgtime = trans_time/iterations;
    auto bytes = (size_t)order * (size_t)order * sizeof(float);
    std::cout << "Rate (MB/s): " << 1.0e-6 * (2L*bytes)/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  } else {
#ifdef VERBOSE
    for (auto i=0; i<order; i++) {
      for (auto j=0; j<order; j++) {
        std::cout << "(" << i << "," << j << ") = " << h_a[i*order+j] << ", " << h_b[i*order+j] << "\n";
      }
    }
#endif
    std::cout << "ERROR: Aggregate squared error " << abserr
              << " exceeds threshold " << epsilon << std::endl;
    return 1;
  }

  prk::CUDAcheck( hipHostFree(h_b) );

  return 0;
}


